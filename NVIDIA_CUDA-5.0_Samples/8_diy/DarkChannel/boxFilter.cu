#include "hip/hip_runtime.h"

#include "boxFilter.cuh"
#include <hip/hip_runtime.h>

#define		BLOCK_SIZE_1D	64	// BLOCK_SIZE = BLOCK_SIZE_1D

__global__ void scan_kernel( float *imDst_C,float * imSrc_C, int nElement, int nBlockStride, int nThreadStride )
{
	__shared__ float sdata[BLOCK_SIZE_1D*2];//blockDim.x*2
	int offset = 0;
	for(  ; offset < nElement ; offset += blockDim.x )
	{
		int length = blockDim.x;
		if( offset + blockDim.x > nElement )
		{
			length = nElement % blockDim.x;
		}

		if( threadIdx.x+offset >= nElement )
			return;

		sdata[threadIdx.x] = imSrc_C[(threadIdx.x+offset)*nThreadStride + blockIdx.x*nBlockStride ];

		if( offset!=0 && threadIdx.x == 0 )
			sdata[0] += imDst_C[(threadIdx.x+offset-1)*nThreadStride + blockIdx.x*nBlockStride ];

		__syncthreads();

		int first = 0;

		for ( int d=1;d<length; d+=d, first=blockDim.x-first )
		{
			if( threadIdx.x < d )
				sdata[threadIdx.x+blockDim.x-first] = sdata[threadIdx.x+first];
			else
				sdata[threadIdx.x+blockDim.x-first] = sdata[threadIdx.x+first] + sdata[threadIdx.x-d+first];
			__syncthreads();
		}

		imDst_C[(threadIdx.x+offset)*nThreadStride + blockIdx.x*nBlockStride ] = sdata[threadIdx.x+first];
		__syncthreads();

	}
}

__global__ void delta( float *imDst_C,float *imCum_C,int r, int nElement, int nBlockStride, int nThreadStride)
{
	if( threadIdx.x > nElement )
		return;

	for( int nCurrentIndex = threadIdx.x;nCurrentIndex< nElement; nCurrentIndex+=blockDim.x )
	{
		int nCurrentElement = nCurrentIndex*nThreadStride + blockIdx.x*nBlockStride;

		if( nCurrentIndex < r + 1 )
			imDst_C[ nCurrentElement ] =	imCum_C[ nCurrentElement + r*nThreadStride ];
		else if ( nCurrentIndex >= r + 1 && nCurrentIndex < nElement-r )
			imDst_C[ nCurrentElement ] =	imCum_C[ nCurrentElement + r*nThreadStride ] -  
															imCum_C[ nCurrentElement - (r + 1)*nThreadStride ] ;
		else //if ( nCurrentIndex >= height-r && nCurrentIndex < height )
			imDst_C[ nCurrentElement ] =	imCum_C[ (nElement-1)*nThreadStride + blockIdx.x*nBlockStride ] -  
															imCum_C[ nCurrentElement - (r + 1)*nThreadStride ] ;
	}

}

//boxfilter
/*%   BOXFILTER   O(1) time box filtering using cumulative sum
%
%   - Definition imDst(x, y)=sum(sum(imSrc(x-r:x+r,y-r:y+r)));
%   - Running time independent of r; 
%   - Equivalent to the function: colfilt(imSrc, [2*r+1, 2*r+1], 'sliding', @sum);
%   - But much faster.*/
void boxfilter0(float *imSrc,float *imCum_C,float *imDst,int r,int height,int width)
{
	
	int nBlockSize;

	// ��ֱY�����ۼӣ��뾶r����2*r+1�����ۼ�
	nBlockSize = height>BLOCK_SIZE_1D? BLOCK_SIZE_1D:  height;

	scan_kernel<<<width, nBlockSize >>>( imCum_C,imSrc, height, 1, width);// ��ֱY�����ۼ�

	delta<<<width,nBlockSize>>>( imDst,imCum_C,r, height, 1, width ); // ��ֱY����Ⱦ������

	// ˮƽX�����ۼӣ��뾶r����2*r+1�����ۼ�
	nBlockSize = width>BLOCK_SIZE_1D? BLOCK_SIZE_1D: width ;

	scan_kernel<<<height, nBlockSize >>>( imCum_C,imDst, width, width, 1);// ˮƽX�����ۼ�

	delta<<<height,nBlockSize>>>( imDst,imCum_C,r, width, width, 1 ); // ˮƽX����Ⱦ������
}



// process row
__device__ void
d_boxfilter_x(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);

    float t;
    // do left edge
    t = id[0] * r;

    for (int x = 0; x < (r + 1); x++)
    {
        t += id[x];
    }

    od[0] = t * scale;

    for (int x = 1; x < (r + 1); x++)
    {
        t += id[x + r];
        t -= id[0];
        od[x] = t * scale;
    }

    // main loop
    for (int x = (r + 1); x < w - r; x++)
    {
        t += id[x + r];
        t -= id[x - r - 1];
        od[x] = t * scale;
    }

    // do right edge
    for (int x = w - r; x < w; x++)
    {
        t += id[w - 1];
        t -= id[x - r - 1];
        od[x] = t * scale;
    }
}

// process column
__device__ void
d_boxfilter_y(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);

    float t;
    // do left edge
    t = id[0] * r;

    for (int y = 0; y < (r + 1); y++)
    {
        t += id[y * w];
    }

    od[0] = t * scale;

    for (int y = 1; y < (r + 1); y++)
    {
        t += id[(y + r) * w];
        t -= id[0];
        od[y * w] = t * scale;
    }

    // main loop
    for (int y = (r + 1); y < (h - r); y++)
    {
        t += id[(y + r) * w];
        t -= id[((y - r) * w) - w];
        od[y * w] = t * scale;
    }

    // do right edge
    for (int y = h - r; y < h; y++)
    {
        t += id[(h-1) * w];
        t -= id[((y - r) * w) - w];
        od[y * w] = t * scale;
    }
}

__global__ void
d_boxfilter_x_global(float *id, float *od, int w, int h, int r)
{
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_x(&id[y * w], &od[y * w], w, h, r);
}

__global__ void
d_boxfilter_y_global(float *id, float *od, int w, int h, int r)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_y(&id[x], &od[x], w, h, r);
}

#if 0
// texture version
// texture fetches automatically clamp to edge of image
__global__ void
d_boxfilter_x_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int x =- r; x <= r; x++)
    {
        t += tex2D(tex, x, y);
    }

    od[y * w] = t * scale;

    for (int x = 1; x < w; x++)
    {
        t += tex2D(tex, x + r, y);
        t -= tex2D(tex, x - r - 1, y);
        od[y * w + x] = t * scale;
    }
}

__global__ void
d_boxfilter_y_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int y = -r; y <= r; y++)
    {
        t += tex2D(tex, x, y);
    }

    od[x] = t * scale;

    for (int y = 1; y < h; y++)
    {
        t += tex2D(tex, x, y + r);
        t -= tex2D(tex, x, y - r - 1);
        od[y * w + x] = t * scale;
    }
}
#endif

void boxfilter(float *imSrc,float *imCum_C,float *imDst,int r,int height,int width)
{
	d_boxfilter_x_global<<< height / BLOCK_SIZE_1D, BLOCK_SIZE_1D, 0 >>>(imSrc, imCum_C, width, height, r);
	d_boxfilter_y_global<<< width / BLOCK_SIZE_1D, BLOCK_SIZE_1D, 0 >>>(imCum_C, imDst, width, height, r);

}
