#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Inverse Matrix by CUBLAS library
8.3. LU�ֽ� cublas<t>getrfBatched(): http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getrfbatched  in cuda5.0
8.4. ����   cublas<t>getriBatched(): http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-getribatched  in cuda5.5

10.2. ϡ�����LU�ֽ� cusparse<t>csrilu0 : http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrilu0

*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes CUDA BLAS
#include <hipblas.h>
#include <hipsparse.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include "matrixInverseBLAS.cuh"

#define BENCH_MATRIX_EXP			7 //2~10
#define BENCH_MATRIX_ROWS           (1<<BENCH_MATRIX_EXP)
#define CUBLAS_TEST_COUNT			(1) // 10~1000


#define SWITCH_CHAR             '-'

__inline__ __device__ __host__  float cuGet(double x)
{
    return float(x);
}


extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}


int verifyResult( T_ELEM *A, T_ELEM *B , int n ) 
{
    T_ELEM *C  = (T_ELEM *)malloc( n * n * sizeof(T_ELEM));

	/* Host implementation of a simple version of sgemm */
    int i;
    int j;
    int k;

    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < n; ++j)
        {
            T_ELEM prod = 0;

            for (k = 0; k < n; ++k)
            {
                prod += A[k * n + i] * B[j * n + k];
            }

            C[j * n + i] = prod;
        }
    }

	for (i = 0; i < n; ++i)
    {
		if( fabs( C[i * n + i] - 1.0f ) > 1.0e-3 )
		{
			free(C);
			return i;
		}
	}

	free(C);
	return 0;
}

int verifyResultBLAS( T_ELEM *A, T_ELEM *B , int n ) 
{
    hipblasStatus_t status;
	
	// blas config
    hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = 1.0f;
	float beta = 0.0f;

	T_ELEM *d_C;
	hipMalloc((void **)&d_C, n * n * sizeof(T_ELEM));
    T_ELEM *C  = (T_ELEM *)malloc( n * n * sizeof(T_ELEM));

    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, d_C, n);

	hipMemcpy( C, d_C, n * n * sizeof(T_ELEM) , hipMemcpyDeviceToHost );


	hipFree( d_C );
	hipblasDestroy(handle);

	for (int i = 0; i < n; ++i)
    {
		if( fabs( C[i * n + i] - 1.0f ) > 1.0e-3 )
		{
			free(C);
			return i;
		}
	}

	free(C);
	return 0;

}

// LU �ֽ⣬ϡ�����
int luDecomposeSparse( T_ELEM **devPtrA , int n )
{
	
	 /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
	
	hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	// get element count of nonzero
	int nz = 0;
	int *nnzPerRow = NULL;
	hipMalloc( (void**)&nnzPerRow,  n*sizeof(int) );
	hipsparseSnnz( cusparseHandle, HIPSPARSE_DIRECTION_ROW, n, 
             n, descr, 
             devPtrA[0], 
             n, nnzPerRow, &nz );

	// sparse matrix
	int *d_col, *d_row;
    float *d_val;
	checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (n+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));

	cusparseStatus = hipsparseSdense2csr( cusparseHandle, n, 
             n, descr, 
             devPtrA[0], n, 
			 nnzPerRow, 
             d_val, d_row, d_col) ;
	
	if (cusparseStatus != HIPBLAS_STATUS_SUCCESS)
    {
          hipError_t cuError = hipGetLastError();
          fprintf(stderr, "!!!! GPU program execution error : cublas Error=%d, cuda Error=%d,(%s)\n", cusparseStatus, cuError,hipGetErrorString(cuError));
          return -1;
    }

	return 0;
}

// �����������棬����blas�⣬ C[i] = A[i] ^ -1
// inverse batch of matrices
int inverseMatrixBLAS( T_ELEM **A , T_ELEM **C , int matrixRows , int sizeBatch ,int bDebug )
{
	int  matrixSize = matrixRows * matrixRows;
	hipError_t err1;
	hipblasStatus_t status;

	// temp data
    T_ELEM **devPtrA = 0;
    T_ELEM **devPtrA_dev = NULL;

	T_ELEM **devPtrC = 0;
    T_ELEM **devPtrC_dev = NULL;

	// temp data for matrix A, input matrix
    devPtrA =(T_ELEM **)malloc( sizeBatch * sizeof(T_ELEM));
	for (int i = 0; i < sizeBatch ; i++)
    {
        hipMalloc((void **)&devPtrA[i], matrixSize * sizeof(T_ELEM));
		hipblasSetMatrix( matrixRows, matrixRows, sizeof(T_ELEM), A[i], matrixRows, devPtrA[i], matrixRows);
	}
	
    hipMalloc((void **)&devPtrA_dev, sizeBatch * sizeof(T_ELEM));
    hipMemcpy(devPtrA_dev, devPtrA, sizeBatch * sizeof(*devPtrA), hipMemcpyHostToDevice);


	// temp data for matrix C, output inverse matrix of A
	devPtrC =(T_ELEM **)malloc( sizeBatch * sizeof(T_ELEM));
	for (int i = 0; i < sizeBatch ; i++)
    {
        hipMalloc((void **)&devPtrC[i], matrixSize * sizeof(T_ELEM));
	}

    hipMalloc((void **)&devPtrC_dev, sizeBatch * sizeof(T_ELEM));
    hipMemcpy(devPtrC_dev, devPtrC, sizeBatch * sizeof(*devPtrC), hipMemcpyHostToDevice);


	// temp data middle
	int *d_pivotArray = NULL;
	int *d_infoArray = NULL;
	hipMalloc( (void**)&d_pivotArray, matrixRows*sizeBatch*sizeof(int) );
	hipMalloc( (void**)&d_infoArray,  sizeBatch*sizeof(int) );
	
	int *h_infoArray = NULL;

	// blas config
    hipblasHandle_t handle;
	hipblasCreate(&handle);
    hipblasSetStream(handle, 0 );

	// timer begin
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

#if 0
	luDecomposeSparse( devPtrA, matrixRows );

	//d_pivotArray[];
	//d_infoArray[];

#else
	// LU factorization �� ����LU���Ƿֽ�
	status = hipblasSgetrfBatched(handle, 
		matrixRows, 
		devPtrA_dev, 
		matrixRows,
		d_pivotArray,
		d_infoArray,
		sizeBatch);

	
	if (status != HIPBLAS_STATUS_SUCCESS)
    {
          hipError_t cuError = hipGetLastError();
          fprintf(stderr, "!!!! GPU program execution error : cublas Error=%d, cuda Error=%d,(%s)\n", status, cuError,hipGetErrorString(cuError));
          return -1;
    }
#endif
	// ���LU�ֽ��Ƿ�˳��ִ��
	if( bDebug )
	{
		h_infoArray = (int*)malloc( sizeBatch*sizeof(int) );

		hipMemcpy( h_infoArray,  d_infoArray, sizeBatch*sizeof(int), hipMemcpyDeviceToHost );

		for(int i=0;i<sizeBatch;i++)
		{
			if( h_infoArray[i] == 0 )
			{
				//fprintf(stderr, "%d-th matrix lu-decompose successed, !\n", i );
				continue;
			}
			else if (h_infoArray[i] > 0)
			{
				fprintf(stderr, "%d-th matrix lu-decompose failed, U(%d,%d) = 0!\n", i, h_infoArray[i], h_infoArray[i] );
				continue;
			}
			else
			{
				fprintf(stderr, "%d-th matrix lu-decompose failed, the %d-th parameter had an illegal value!\n", i, -h_infoArray[i] );
				continue;
			}
		}
	}

#if 1// if 0 ��������һ��LU�ֽ��ʱ�䣬 1536*1536����ʱ��ֲ���700-1800ms GPU480
	// inversion of matrices A, output result to matrices C �� ���Ǿ�������
	status = hipblasSgetriBatched(handle, 
		matrixRows, 
		devPtrA_dev, 
		matrixRows,
		d_pivotArray,
		devPtrC_dev,
		matrixRows,
		d_infoArray,
		sizeBatch);

	if (status != HIPBLAS_STATUS_SUCCESS)
    {
          hipError_t cuError = hipGetLastError();
          fprintf(stderr, "!!!! GPU program execution error : cublas Error=%d, cuda Error=%d,(%s)\n", status, cuError,hipGetErrorString(cuError));
          return -1 ;
    }
#endif
	// ������Ǿ��������Ƿ�˳��ִ��
	if( bDebug )
	{
		hipMemcpy( h_infoArray,  d_infoArray, sizeBatch*sizeof(int), hipMemcpyDeviceToHost );

		for(int i=0;i<sizeBatch;i++)
		{
			if( h_infoArray[i] == 0 )
			{
				//fprintf(stderr, "%d-th matrix lu-decompose successed, !\n", i );
				continue;
			}
			else if (h_infoArray[i] > 0)
			{
				fprintf(stderr, "%d-th matrix lu-decompose failed, U(%d,%d) = 0!\n", i, h_infoArray[i], h_infoArray[i] );
				continue;
			}
		}
	}

	// timer end
	hipError_t cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
    {
		fprintf(stderr, "!!!! GPU program execution error on hipDeviceSynchronize : hipError_t=%d,(%s)\n", cudaStatus,hipGetErrorString(cudaStatus));
		return -1;
	}

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

	// ����������Դ淵���ڴ棬gpu -> cpu
	for(int i=0; i< sizeBatch; i++)
	{
		hipMemcpy( C[i], devPtrC[i], matrixSize * sizeof(T_ELEM) , hipMemcpyDeviceToHost );
	}

	// ��֤�����������Ƿ���ȷ
	if( bDebug )
	{
		int bStatus = 0;
		for(int i=0; i< sizeBatch; i++)
		{
	#if 0
			bStatus = verifyResultBLAS( devPtrA[i], devPtrC[i], matrixRows );
	#else
			bStatus = verifyResult( A[i], C[i], matrixRows );	
	#endif
			if( bStatus )
			{
				printf( "Matrix Inverse Wrong! A*A^(-1) [%d,%d] !=1 \n" ,bStatus ,bStatus );
				break;
			}
		}
	}


    // �ͷŸ��������õ�����ʱ�ڴ�
	for(int i = 0; i < sizeBatch; ++i) 
	{       
            if(devPtrA[i]) hipFree(devPtrA[i]);
            if(devPtrC[i]) hipFree(devPtrC[i]);
	}  

	if (devPtrA) free(devPtrA);           
	if (devPtrC) free(devPtrC); 

	if (devPtrA_dev)	hipFree(devPtrA_dev);
	if (devPtrC_dev)	hipFree(devPtrC_dev); 

	if (d_pivotArray)	hipFree(d_pivotArray);
	if (d_infoArray)	hipFree(d_infoArray); 
	if (h_infoArray)	free(h_infoArray); 

	hipblasDestroy(handle);

	return 0;
}

// �����������棬����blas�⣬ C = A ^ -1
// inverse a matrix
int inverseMatrixBLAS( T_ELEM *A , T_ELEM *C, int matrixRows, int bDebug )
{
	// ��ʼ��matrix A, input matrix
	T_ELEM **ABatch  = (T_ELEM **)malloc( 1 * sizeof(T_ELEM*));
	*ABatch  = A;

	// matrix C, output inverse matrix of A
	T_ELEM **CBatch  = (T_ELEM **)malloc( 1 * sizeof(T_ELEM*));
	*CBatch  = C;

	inverseMatrixBLAS( ABatch, CBatch, matrixRows, 1, bDebug ) ;

	if (ABatch) free (ABatch);
    if (CBatch) free (CBatch);

	return 0;
}

